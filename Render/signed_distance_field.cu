#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <glm/glm.hpp>
#include <limits>
#include "signed_distance_field.h"`
#include "graphics_lib/Utilities/cuda_helper.h"

__host__ __device__
float triangle_p_distance (vec3 p0, vec3 p1, vec3 p2, vec3 p) {
	// project p into the plane
	vec3 n = glm::normalize(glm::cross(p1 - p0, p2 - p0));
	float proj_length = glm::dot((p - p0), n);
	vec3 pp = p - proj_length * n;

	mat3 bary_mat(p0, p1, p2);
	if (glm::determinant(bary_mat) > 0) {
		vec3 bary_coord = glm::inverse(bary_mat) * pp;
		if (bary_coord.x >= 0 && bary_coord.x <= 1.0 &&
			bary_coord.y >= 0 && bary_coord.y <= 1.0 &&
			bary_coord.z >= 0.0 && bary_coord.z <= 1.0) {
			return std::abs(proj_length);
		}
	}
	
	float ret = FLT_MAX;
	float p0_dis = glm::distance(p0, p);
	float p1_dis = glm::distance(p1, p);
	float p2_dis = glm::distance(p2, p);
	if (p0_dis < ret) ret = p0_dis;
	if (p1_dis < ret) ret = p1_dis;
	if (p2_dis < ret) ret = p2_dis;

	return ret;
};

__host__ __device__
float signed_distance(vec3 *verts, int N, vec3 p) {
	float mini_dis = FLT_MAX;
	float winding_sum = 0.0f;

	for (int ti = 0; ti < N / 3; ++ti) {
		//printf("%d %d \n", ti, N / 3);
		vec3 p0 = verts[3 * ti + 0];
		vec3 p1 = verts[3 * ti + 1];
		vec3 p2 = verts[3 * ti + 2];
		float cur_dis = triangle_p_distance(p0, p1, p2, p);
		if (cur_dis < mini_dis) {
			mini_dis = cur_dis;
		}
	}
	// todo, winding sum to decide inside or outside
	return mini_dis;
}

__global__
void cuda_construct(glm::vec3 *verts, int N, float *sdf, int sdf_N, int w, int h, int z, AABB field_range) {
	vec3 diag = field_range.diagonal();
	vec3 p0 = field_range.p0;
	
	int ind = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	
	for (int sdfi = ind; sdfi < sdf_N; sdfi += stride) {
		int k = sdfi / (w * h);
		int j = (sdfi - (w*h*k)) / w;
		int i = sdfi - (w * h * k) - j * w;
		vec3 p = p0 + diag * vec3((float)i / w, (float)j / h, (float)k / z);
		sdf[sdfi] = signed_distance(verts, N, p);
		//sdf[sdfi] = (float)i + (float)j + (float)k;
	}
}

signed_distance_field::signed_distance_field(int w, int h, int z):m_w(w), m_h(h), m_z(z){
	m_field_buffer.resize(w * h * z);
}

void signed_distance_field::construct(std::shared_ptr<mesh> m, AABB field_range) {
	m_field_range = field_range;
	
	cuda_timer clc;
	clc.tic();
	// memory
	container_cuda<vec3> d_verts(m->m_verts);
	container_cuda<float> d_sdf(m_field_buffer);

	int grid = 512, block = (d_sdf.get_n() + grid - 1) / grid;
	cuda_construct<<<grid, block >>>(d_verts.get_d(), d_verts.get_n(), d_sdf.get_d(), d_sdf.get_n(), m_w, m_h, m_z, m_field_range);
	GC(hipDeviceSynchronize());

	// copy back
	d_sdf.mem_copy_back();
	clc.toc();
	INFO("constructing spent: " + std::to_string(clc.get_time()) + "ms");

	// CPU version
	//vec3 diag = m_field_range.diagonal();
	//for (int i = 0; i < m_w; ++i) {
	//	for (int j = 0; j < m_h; ++j) {
	//		for (int k = 0; k < m_z; ++k) {
	//			// compute closest point
	//			vec3 p = (vec3((float)i / m_w, (float)j / m_h, (float)k / m_z) + vec3(0.5f)) * diag + m_field_range.p0;
	//			float dis = close_distance(m, p);
	//			at(i, j, k) = dis;
	//		}
	//	}
	//}
}

void signed_distance_field::to_visualize_mesh(std::shared_ptr<mesh> m) {
	m->clear_vertices();
	float max_value = 0.0f;
	vec3 diag = m_field_range.diagonal();
	for (int i = 0; i < m_w; ++i) {
		for (int j = 0; j < m_h; ++j) {
			for (int k = 0; k < m_z; ++k) {
				float value = at(i, j, k);
				vec3 p = (vec3((float)i / m_w, (float)j / m_h, (float)k / m_z)) * diag;
				vec3 p2 = (vec3((float)(i+1) / m_w, (float)(j+1) / m_h, (float)(k+1) / m_z)) * diag;
				AABB cur_aabb(p, p2);
				auto cur_verts = cur_aabb.to_tri_mesh();
				m->m_verts.insert(m->m_verts.end(), cur_verts.begin(), cur_verts.end());
				for (int i = 0; i < cur_verts.size(); ++i) {
					m->m_colors.push_back(glm::vec3(value));
				}
				max_value = std::max(max_value, value);
			}
		}
	}

	for (auto &c : m->m_colors) {
		c = c / max_value;
	}
}

bool signed_distance_field::query(glm::vec3 p, float &ret) {
	vec3 relative = p - m_field_range.p0;
	vec3 diag = m_field_range.diagonal();
	vec3 uvw = relative / diag;

	// outside of the volume 
	if (uvw.x < 0 || uvw.x >1.0 || uvw.y < 0 || uvw.y >1.0 || uvw.z < 0 || uvw.z >1.0) {
		return false;
	}

	float x = uvw.x * (m_w - 1);
	float y = uvw.y * (m_h - 1);
	float z = uvw.z * (m_z - 1);
	// Tri-linear interpolation
	float p0, p1, p2, p3, p4, p5, p6, p7, p8;
	int centerx = (int)x, centery = (int)y, centerz = (int)z;
	p0 = at(centerx, centery, centerz);
	p1 = at(centerx + 1, centery, centerz);
	p2 = at(centerx, centery, centerz + 1);
	p3 = at(centerx + 1, centery, centerz + 1);

	p4 = at(centerx, centery+1, centerz);
	p5 = at(centerx + 1, centery+1, centerz);
	p6 = at(centerx, centery+1, centerz + 1);
	p7 = at(centerx + 1, centery+1, centerz + 1);

	// a-b
	// c-d
	auto bilinear = [&](float a, float b, float c, float d, glm::vec2 t) {
		glm::mat2 w;
		w[0] = glm::vec2(a, b);
		w[1] = glm::vec2(c, d);
		w = glm::transpose(w);
		return glm::dot(vec2(1.0 - t.y, t.y), w * vec2(1.0 - t.x, t.x));
	};
	float d1 = bilinear(p0, p1, p2, p3, glm::vec2(uvw.x, uvw.z));
	float d2 = bilinear(p4, p5, p6, p7, glm::vec2(uvw.x, uvw.z));
	ret = pd::lerp(d1, d2, uvw.y);
}

float& signed_distance_field::at(int u, int v, int w) {
	u = pd::clamp(u, 0, m_w-1);
	v = pd::clamp(v, 0, m_h-1);
	w = pd::clamp(w, 0, m_z-1);

	return m_field_buffer[u * m_h * m_z + v * m_z + w];
}

float signed_distance_field::close_distance(std::shared_ptr<mesh> m, glm::vec3 p) {
	float ret = std::numeric_limits<float>::max();
	if (!m) {
		return ret;
	}

	auto world_verts = m->compute_world_space_coords();
	for (int ti = 0; ti < world_verts.size()/3; ++ti) {
		vec3 p0 = world_verts[3 * ti + 0];
		vec3 p1 = world_verts[3 * ti + 1];
		vec3 p2 = world_verts[3 * ti + 2];
		
		float dis = triangle_p_distance(p0, p1, p2, p);
		ret = std::min(ret, dis);
	}
	return ret;
}
